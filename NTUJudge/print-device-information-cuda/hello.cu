#include <stdio.h>

int main(){
	int nDevices;

	hipGetDeviceCount(&nDevices);

	printf("%d devices found supporting CUDA\n", nDevices);
	for(int i = 0; i < nDevices; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("----------------------------------\n");
		printf("Device %s\n", prop.name);
		printf("----------------------------------\n");
		printf(" Device memory:     %zu\n", prop.totalGlobalMem);
		printf(" Memory per-block:     %zu\n", prop.sharedMemPerBlock);
		printf(" Register per-block:     %d\n", prop.regsPerBlock);
		printf(" Warp size:         %d\n", prop.warpSize);
		printf(" Memory pitch:         %zu\n", prop.memPitch);
		printf(" Constant Memory:     %zu\n", prop.totalConstMem);
		printf(" Max thread per-block:     %d\n", prop.maxThreadsPerBlock);
		printf(" Max thread dim:     %d / %d / %d\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf(" Max grid size:     %d / %d / %d\n", prop.maxGridSize[0],
			prop.maxGridSize[1], prop.maxGridSize[2]);
		printf(" Ver:             %d.%d\n", prop.major, prop.minor);
		printf(" Clock:         %d\n", prop.clockRate);
		printf(" Texture Alignment:     %zu\n", prop.textureAlignment);
	}
}