#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include <inttypes.h>
#include <stdint.h>
// #include "utils.h"
#include <hip/hip_runtime.h>
#include <math.h>
 
#define MAXGPU 8
#define MAXCODESZ 32767
#define MAXN 16777216

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
__device__ uint32_t rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}

__device__ uint32_t encrypt(uint32_t m, uint32_t key) {
    return (rotate_left(m, key&31) + key)^key;
}

__global__ void dot(uint32_t key1, uint32_t key2, uint32_t C[MAXN], int N){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ uint32_t local_C[512];    
    int bid = blockDim.x / 2;
    int A, B;
    uint32_t temp = 0;
    while(idx < N){
        A = encrypt(idx, key1);
        B = encrypt(idx, key2);
        temp += A * B;
        idx += gridDim.x * blockDim.x;
    }
    local_C[threadIdx.x] = temp;
    __syncthreads();
    while(bid > 0){
        if(threadIdx.x < bid){
            local_C[threadIdx.x] += local_C[threadIdx.x+bid];
        }
        bid /= 2;
        __syncthreads();
    }
    if(threadIdx.x == 0){
        C[blockIdx.x] = local_C[0];
    }
}


uint32_t A[MAXN], B[MAXN], C[MAXN];
int main(int argc, char *argv[]) {
    uint32_t *device_A, *device_B, *device_C;
    int num_thread = 512;
    int num_block = 1024;
    // printf("declare");
    // HANDLE_ERROR(hipMalloc((void **)&device_A, sizeof(uint32_t) * MAXN));
    // HANDLE_ERROR(hipMalloc((void **)&device_B, sizeof(uint32_t) * MAXN));
    HANDLE_ERROR(hipMalloc((void **)&device_C, sizeof(uint32_t) * ceil(MAXN / num_thread)));
    // printf("error");
    // omp_set_num_threads(4);
    int N;
    uint32_t key1, key2;
    while (scanf("%d %" PRIu32 " %" PRIu32, &N, &key1, &key2) == 3) {
        // int chunk = N / 4;
        // printf("while %d", N);
// #pragma omp parallel for schedule(static, chunk)
//         for (int i = 0; i < N; i++) {
//             A[i] = encrypt(i, key1);
//             B[i] = encrypt(i, key2);
//         }
 
        // HANDLE_ERROR(hipMemcpy(device_A, A, sizeof(uint32_t) * MAXN,hipMemcpyHostToDevice));
        // HANDLE_ERROR(hipMemcpy(device_B, B, sizeof(uint32_t) * MAXN,hipMemcpyHostToDevice));
        

        dot <<<min(num_block, int(ceil(N / num_thread))), num_thread>>> (key1, key2, device_C, N);

        HANDLE_ERROR(hipMemcpy(C, device_C, sizeof(uint32_t) * min(num_block, int(ceil(N / num_thread))), hipMemcpyDeviceToHost));

        // for (int i = 0; i < N; i++)
        //     C[i] = A[i] * B[i];
 
        uint32_t sum = 0;
#pragma omp parallel for reduction(+ : sum)
        for (int i = 0; i < min(num_block, int(ceil(N / num_thread))); i++)
            sum += C[i];
        printf("%" PRIu32 "\n", sum);
    }

    hipFree(device_C);

    return 0;
}